// include any headers
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// define constants
#define RADIUS 3
#define BLOCK_SIZE 256

// function to apply a 1d stencil to an array
__global__ void stencil_1d(int *in, int *out, int len) {
  __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
  int gindex = threadIdx.x + blockIdx.x * blockDim.x;
  int lindex = threadIdx.x + RADIUS;

  temp[lindex] = in[gindex];
  if (threadIdx.x < RADIUS){ //checks if lower than radius, aka will go off end
    
    if(gindex-RADIUS < 0){
      temp[lindex-RADIUS]=0;
    }
    else{

    temp[lindex - RADIUS] = in[gindex - RADIUS];
    }
    // I think this fixes boundary problem...?
    if (gindex + BLOCK_SIZE < len) {
        temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE]; // Right neighbor
    } else {
        temp[lindex + BLOCK_SIZE] = 0; // Padding for out-of-bounds (right side)
    }
  //   if(gindex > len){ //WRONG boooo :() padding to the right wrong
  //     temp[lindex]=0;
  //   }
  //   else{
  //   temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
  // }
  }

  //test statement

  __syncthreads();
  //apply stencil I think....????
  int result = 0;
  for(int offset =- RADIUS ; offset <= RADIUS ; offset++)
    result += temp[lindex + offset];

  out[gindex] = result;

}

int main(void) {
  // allocate input and output arrays
  int N = 1<<20; // 1M elements
  N=526;
  int *input, *output;
  hipMallocManaged(&input, N*sizeof(int));
  hipMallocManaged(&output, N*sizeof(int));

  // initialize input array on the host
  for(int i=0; i<N; i++){
    //if all ones
    input[i]=1;
    printf("%d ", input[i]);
    
  }
  printf("\n");

  // run the kernel
  //Initializing block size and running kernerl
  int blocksPerGrid = (N + BLOCK_SIZE - 1) / BLOCK_SIZE; 
  int threadsPerBlock = BLOCK_SIZE;
  

  stencil_1d<<<blocksPerGrid, threadsPerBlock>>>(input, output, N);
  hipDeviceSynchronize();


  // check results
  for (int i = 0; i < N; i++) {
    printf("%d ", output[i]);
  }
  printf("\n");

  printf("%s\n", hipGetErrorString(hipGetLastError()));

  // Check for errors (from add, where all values should be 3.0f)
  // float maxError = 0.0f;
  // for (int i = 0; i < N; i++){
  //   maxError = fmax(maxError, fabs(y[i]-3.0f));
  // std::cout << "Max error: " << maxError << std::endl;
  // }

  // free memory
  hipFree(input);
  hipFree(output);

  return 0;
}