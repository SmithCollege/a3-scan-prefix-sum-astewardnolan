// include any headers
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

// define constants
#define BLOCK_SIZE 128

__global__ void scan(int *in, int *out, int len) { //need kernel  function
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;

    if(gindex >= len){
        return;
    }

    for (int i = 0; i < len; i++) {
        int value = 0;
        for (int j = 0; j <= i; j++) {
            value += in[j];
   }
    out[i] = value;
  }
    __syncthreads();
}

double get_clock() {
    struct timeval tv; int ok;
    ok = gettimeofday(&tv, (void *) 0);
    if (ok<0) { 
        printf("gettimeofday error"); 
        }
    return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

int main(void) {
  // allocate input and output arrays
  int SIZE = 1<<20; // 1M elements
  SIZE=128;
  int *input, *output;
  hipMallocManaged(&input, SIZE*sizeof(int));
  hipMallocManaged(&output, SIZE*sizeof(int));

  // initialize inputs
  for (int i = 0; i < SIZE; i++) {
    input[i] = 1;
   }
    double start = get_clock();
//checks og
  printf("\n");

      // run the kernel
  //Initializing block size and running kernerl
  int blocksPerGrid = (SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE; 
  int threadsPerBlock = BLOCK_SIZE;


  scan<<<blocksPerGrid, threadsPerBlock>>>(input, output, SIZE);
  hipDeviceSynchronize();

  double end = get_clock();
  printf("time per call: %f ns\n", (end-start) );
  // check results
  for (int i = 0; i < SIZE; i++) {
    printf("%d ", output[i]);
  }
  printf("\n");

  printf("%s\n", hipGetErrorString(hipGetLastError()));

  hipFree(input);
  hipFree(output);
  return 0;


}
