#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define BLOCK_SIZE 128

__global__ void recursive_doubling(int *in, int *out, int SIZE) {
    int tIdx = threadIdx.x + blockIdx.x * blockDim.x;

    if (tIdx < SIZE) {
        out[tIdx] = in[tIdx]; 
    }
    __syncthreads(); 

    // recursive doubling part
    for (int offset = 1; offset < SIZE; offset *= 2) {
        if (tIdx >= offset) {
            out[tIdx] += out[tIdx - offset]; // Accumulatez values
        }
        __syncthreads(); 
    }
}

void initialize_data(int *in, int SIZE) {
    for (int i = 0; i < SIZE; ++i) {
        in[i] = i + 1;  
    }
}


double get_clock() {
    struct timeval tv; int ok;
    ok = gettimeofday(&tv, (void *) 0);
    if (ok<0) { 
        printf("gettimeofday error"); 
        }
    return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

int main(void) {
  // allocate input and output arrays
  int SIZE = 1<<20; // 1M elements
  SIZE=128;
  int *in, *out;

  hipMallocManaged(&in, SIZE*sizeof(int));
  hipMallocManaged(&out, SIZE*sizeof(int));

  // initialize inputs
  for (int i = 0; i < SIZE; i++) {
    in[i] = 1;
   }

  initialize_data(in, SIZE); // Initialize data with 1, 2, ..., N

  double start = get_clock();
//checks og
  printf("\n");

      // run the kernel
  //Initializing block size and running kernerl
  int blocksPerGrid = (SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE; 
  int threadsPerBlock = BLOCK_SIZE;


  recursive_doubling<<<blocksPerGrid, threadsPerBlock>>>(in, out, SIZE);
  hipDeviceSynchronize();

  double end = get_clock();
  printf("time per call: %f ns\n", (end-start) );
  // check results
  for (int i = 0; i < SIZE; i++) {
    printf("%d ", out[i]);
  }
  printf("\n");

  printf("%s\n", hipGetErrorString(hipGetLastError()));

  hipFree(in);
  hipFree(out);
  return 0;


}
